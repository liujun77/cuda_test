
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

using namespace std;

__global__ void add(int *a, int *b, int n){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index<n){
        a[index] += b[index];
    }
}

__global__ void rad(int *a, int n){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index<n){
        a[index] = 1;
    }
}

int main(){
    int N = 10000;
    int M = 512;
    int *a, *b;
    int *d_a, *d_b;
    int size = N * sizeof(int);
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);

    a = (int *)malloc(size);
    b = (int *)malloc(size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    rad<<<(N+M-1)/M, M>>>(d_a, size);
    rad<<<(N+M-1)/M, M>>>(d_b, size);
    add<<<(N+M-1)/M, M>>>(d_a, d_b, size);

    hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);
    
    int ret = 0;
    for(int i=0; i<N; i++)
        ret += a[i];
    cout << ret << endl;
    free(a); free(b);
    hipFree(d_a); hipFree(d_b);
    return 0;
}
